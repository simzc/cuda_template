#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_vector_types.h"
#include "MethodDerived.h"
#include ""

template<typename Base, typename Derived>
__global__ void Test() {
    uint tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid >= 1) {
        return;
    }
    
    Base* method = new Derived();
    method->Print();

    delete method;
}

// template<typename MethodBase>
// __global__ void Test(MethodBase* method) {
//     uint tid = blockDim.x * blockIdx.x + threadIdx.x;

//     if(tid >= 10) {
//         return;
//     }
//     method->Print();
// }

int main(int argc, const char** argv) {

    // MethodBase *test = new MethodDerived();
    // MethodBase* test;
    // hipMalloc((void **)&test, sizeof(MethodBase));

    dim3 gridSize(128, 1);
    dim3 blockSize(128, 1);
    Test<MethodBase, MethodDerived><<<gridSize, blockSize>>>();
    hipDeviceSynchronize();

    return 0;
}